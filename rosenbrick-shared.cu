#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <algorithm>
#include <functional>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "constants.h"

// assume block size equal population size

template <class T>
__device__ inline T sqr(const T& value) {
	return value * value;
}

const int MAX_THREADS_PER_BLOCK = 128;

void cudasafe(hipError_t error, char* message = "Error occured") {
	if(error != hipSuccess) {
		fprintf(stderr,"ERROR: %s : %i\n", message, error);
		exit(-1);
	}
}

__global__ void randomInit(hiprandState* state, unsigned long seed) {
    int tid = threadIdx.x;
    hiprand_init(seed, tid, 0, state + tid);
}

__global__ void calcScore(const float* population, ScoreWithId* score) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < POPULATION_SIZE) {
		float result = 0;
	
		const float *curPos = &population[tid * VAR_NUMBER];
		for (size_t i=0; i<VAR_NUMBER-1; ++i) {
			result += sqr(1 - *curPos) + 100 * sqr(*(curPos+1) - sqr(*curPos));
			++curPos;
		}

		score[tid].score = result;
		score[tid].id = tid;
	}
}

struct ScoreCompare {
	__host__ __device__ bool operator() (const ScoreWithId& a, const ScoreWithId& b) const {
		return a.score < b.score;
	}
};

__global__ void produceGeneration(const float* population, float* nextGeneration, const ScoreWithId* score, hiprandState* randomStates) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	float signs[2] = {-1.0f, 1.0f};
	
	float* nextGenerationPos = &nextGeneration[tid * VAR_NUMBER];
	const float* individual = &population[score[tid % (POPULATION_SIZE / 3)].id * VAR_NUMBER];
	//const float* individual = &population[score[tid].id * VAR_NUMBER];

	if (tid < POPULATION_SIZE / 3) { // copy as is
		for (int i=0; i<VAR_NUMBER; ++i) {
			*nextGenerationPos = *individual;
			++nextGenerationPos;
			++individual;
		}
	} else {
		hiprandState &localState = randomStates[threadIdx.x];
		if (tid < POPULATION_SIZE * 2 / 3) { // mutate
			for (int i=0; i<VAR_NUMBER; ++i) {
				const float sign = signs[static_cast<int>(hiprand_uniform(&localState)*2)];
				*nextGenerationPos = *individual + powf(10.0, ((hiprand_uniform(&localState) * 17) - 15)) * sign;
				++nextGenerationPos;
				++individual;
			}
		} else if (tid < POPULATION_SIZE) { // crossover
			const int otherIndividualIndex = (tid + static_cast<int>(hiprand_uniform(&localState) * POPULATION_SIZE)) % (POPULATION_SIZE / 3);
			const float* otherIndividual = &population[otherIndividualIndex * VAR_NUMBER];

			for (int i=0; i<VAR_NUMBER; ++i) {
				*nextGenerationPos = (*individual + *otherIndividual) * 0.5f;
				++nextGenerationPos;
				++individual;
				++otherIndividual;
			}
		}
	}
}

__global__ void GAKernel(float* population, ScoreWithId* score, hiprandState* randomStates) {
	__shared__ float sharedPopulation[MAX_THREADS_PER_BLOCK][VAR_NUMBER];
	__shared__ float sharedScore[MAX_THREADS_PER_BLOCK];
	const float signs[2] = {-1.0f, 1.0f};

	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int tid = threadIdx.x;

	// loading initial random population into shared memory
    if (gid < POPULATION_SIZE) {
        for (int i=0; i<VAR_NUMBER; ++i)
            sharedPopulation[tid][i] = population[gid * VAR_NUMBER + i];
    }

	for (int i=0; i<VAR_NUMBER; ++i)
		sharedPopulation[tid][i] = i + tid;

	hiprandState &localState = randomStates[tid];
	for (int generationIndex=0; ; ++generationIndex) {
		__syncthreads();

		// calculating score
		const float *curPos = sharedPopulation[tid];
		float result = 0;
		for (size_t i=0; i<VAR_NUMBER-1; ++i) {
			result += sqr(1 - *curPos) + 100 * sqr(*(curPos+1) - sqr(*curPos));
			++curPos;
		}
		sharedScore[tid] = result;

		__syncthreads();

		if (generationIndex == 129990) break;

		// selection

		if (tid < MAX_THREADS_PER_BLOCK / 2) {
			// first half of threads writes best individual into its position
			if (sharedScore[tid] > sharedScore[tid + MAX_THREADS_PER_BLOCK / 2]) {
				for (int i=0; i<VAR_NUMBER; ++i)
					sharedPopulation[tid][i] = sharedPopulation[tid + MAX_THREADS_PER_BLOCK / 2][i];
			}
		}

		__syncthreads();

		// now we've got best individuals in the first half of sharedPopulation

		// crossovers
		if (tid > MAX_THREADS_PER_BLOCK / 2) {
			int first = hiprand_uniform(&localState) * (MAX_THREADS_PER_BLOCK / 2);
			int second = hiprand_uniform(&localState) * (MAX_THREADS_PER_BLOCK / 2);
		
			// TODO: implement weight here?
			for (int i=0; i<VAR_NUMBER; ++i) {
				sharedPopulation[tid][i] = (sharedPopulation[first][i] + sharedPopulation[second][i]) * 0.5f;
			}
		}

		__syncthreads();

		// mutations
        if (tid > MAX_THREADS_PER_BLOCK / 2) {
            if (hiprand_uniform(&localState) < 0.8) {
                for (int i=0; i<VAR_NUMBER; ++i) {
                    const float sign = signs[static_cast<int>(hiprand_uniform(&localState)*2)];
                    sharedPopulation[tid][i] += powf(10.0, ((hiprand_uniform(&localState) * 17) - 15)) * sign;
                }
            }
        }
	}

	// output current population back
    if (gid < POPULATION_SIZE) {
        for (int i=0; i<VAR_NUMBER; ++i)
            population[gid * VAR_NUMBER + i] = sharedPopulation[tid][i];

        score[gid].score = sharedScore[tid];
    }
}

void printPopulation(const float* devicePopulation, const ScoreWithId* deviceScore) {
	float population[POPULATION_SIZE][VAR_NUMBER];
	cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

	ScoreWithId score[POPULATION_SIZE];
	cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");

	//std::cout.cetf(std::ios::fixed);
	std::cout.precision(12);
	
	for (int i=0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << i << ' ';
	}
	std::cout << std::endl;

	for (int i=0; i<VAR_NUMBER; i++) {
		for (int u=0; u<POPULATION_SIZE; ++u) {
			std::cout << std::setw(15) << population[u][i] << ' ';
		}
		std::cout << std::endl;
	}
	std::cout << "Score: " << std::endl;
	for (int i=0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << score[i].score << ' ';
	}
	std::cout << std::endl;
}

double solveGPU() {
	double ans = 0;

	float *population = new float[POPULATION_SIZE * VAR_NUMBER];

	for (int i=0; i<POPULATION_SIZE; ++i) {
		for (int u=0; u<VAR_NUMBER; ++u) {
			population[i * VAR_NUMBER + u] = float_random();
		}
	}

	// copying population to device
	float *devicePopulation = 0;
	float *nextGeneration = 0;
	ScoreWithId *deviceScore = 0;
	hiprandState* randomStates;

	cudasafe(hipMalloc(&randomStates, MAX_THREADS_PER_BLOCK * sizeof(hiprandState)), "Could not allocate memory for randomStates");
	cudasafe(hipMalloc((void **)&devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for devicePopulation");
	cudasafe(hipMalloc((void **)&nextGeneration, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for nextGeneration");
	cudasafe(hipMalloc((void **)&deviceScore, POPULATION_SIZE * sizeof (ScoreWithId)), "Could not allocate memory for deviceScore");

	cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");

	// invoking random init
	randomInit<<<1, MAX_THREADS_PER_BLOCK>>>(randomStates, 900);
	cudasafe(hipGetLastError(), "Could not invoke kernel randomInit");
	cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling randomInit");

	const int BLOCKS_NUMBER = (POPULATION_SIZE + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
	GAKernel<<<BLOCKS_NUMBER, MAX_THREADS_PER_BLOCK>>>(devicePopulation, deviceScore, randomStates);
	cudasafe(hipGetLastError(), "Could not invoke GAKernel");
	cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling GAKernel");

	printPopulation(devicePopulation, deviceScore);

	// freeing memory
	cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");
	cudasafe(hipFree(deviceScore), "Failed to free deviceScore");
	cudasafe(hipFree(randomStates), "Could not free randomStates");
	cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");

	delete[] population;

	return ans;
}

int main() {
	freopen("output.txt", "w", stdout);
	srand(900);
	srand(static_cast<unsigned>(time(0)));

	double ans = solveGPU();
	std::cout << "GPU answer = " << ans << std::endl;

	return 0;
}
