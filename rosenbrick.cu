#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <algorithm>
#include <functional>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "constants.h"

// assume block size equal population size

template <class T>
__device__ inline T sqr(const T& value) {
	return value * value;
}

void cudasafe(hipError_t error, char* message = "Error occured") {
	if(error != hipSuccess) {
		fprintf(stderr,"ERROR: %s : %i\n", message, error);
		exit(-1);
	}
}

__global__ void randomInit(hiprandState* state, unsigned long seed) {
    int tid = threadIdx.x;
    hiprand_init(seed, tid, 0, state + tid);
}

__global__ void calcScore(const float* population, ScoreWithId* score) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < POPULATION_SIZE) {
		float result = 0;
	
		const float *curPos = &population[tid * VAR_NUMBER];
		for (size_t i=0; i<VAR_NUMBER-1; ++i) {
			result += sqr(1 - *curPos) + 100 * sqr(*(curPos+1) - sqr(*curPos));
			++curPos;
		}

		score[tid].score = result;
		score[tid].id = tid;
	}
}

struct ScoreCompare {
	__host__ __device__ bool operator() (const ScoreWithId& a, const ScoreWithId& b) const {
		return a.score < b.score;
	}
};

__global__ void produceGeneration(const float* population, float* nextGeneration, const ScoreWithId* score, hiprandState* randomStates) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	float* nextGenerationPos = &nextGeneration[tid * VAR_NUMBER];
	const float* individual = &population[score[tid % (POPULATION_SIZE / 3)].id * VAR_NUMBER];
	//const float* individual = &population[score[tid].id * VAR_NUMBER];

	if (tid < POPULATION_SIZE / 3) { // copy as is
		for (int i=0; i<VAR_NUMBER; ++i) {
			*nextGenerationPos = *individual;
			++nextGenerationPos;
			++individual;
		}
	} else {
		hiprandState &localState = randomStates[threadIdx.x];
		if (tid < POPULATION_SIZE * 2 / 3) { // mutate
			for (int i=0; i<VAR_NUMBER; ++i) {
				*nextGenerationPos = *individual + powf(10.0, ((hiprand_uniform(&localState) * 17) - 15)) * (hiprand_uniform(&localState) < 0.5f ? -1 : 1);
				++nextGenerationPos;
				++individual;
			}
		} else if (tid < POPULATION_SIZE) { // crossover
			const int otherIndividualIndex = (tid + static_cast<int>(hiprand_uniform(&localState) * POPULATION_SIZE)) % (POPULATION_SIZE / 3);
			const float* otherIndividual = &population[otherIndividualIndex * VAR_NUMBER];

			for (int i=0; i<VAR_NUMBER; ++i) {
				*nextGenerationPos = (*individual + *otherIndividual) * 0.5f;
				++nextGenerationPos;
				++individual;
				++otherIndividual;
			}
		}
	}
}

void printPopulation(const float* devicePopulation, const ScoreWithId* deviceScore) {
	float population[POPULATION_SIZE][VAR_NUMBER];
	cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

	ScoreWithId score[POPULATION_SIZE];
	cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");

	//std::cout.cetf(std::ios::fixed);
	std::cout.precision(12);
	for (int i=0; i<VAR_NUMBER; i++) {
		for (int u=0; u<POPULATION_SIZE; ++u) {
			std::cout << std::setw(15) << population[u][i] << ' ';
		}
		std::cout << std::endl;
	}
	for (int i=0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << score[i].score << ' ';
	}
	std::cout << std::endl;
}

double solveGPU() {
	double ans = 0;

	const int MAX_THREADS_PER_BLOCK = 512;

	ScoreWithId score[POPULATION_SIZE];
	float *population = new float[POPULATION_SIZE * VAR_NUMBER];

	for (int i=0; i<POPULATION_SIZE; ++i) {
		for (int u=0; u<VAR_NUMBER; ++u) {
			population[i * VAR_NUMBER + u] = float_random();
		}
	}

	// copying population to device
	float *devicePopulation = 0;
	float *nextGeneration = 0;
	ScoreWithId *deviceScore = 0;
	hiprandState* randomStates;

	cudasafe(hipMalloc(&randomStates, MAX_THREADS_PER_BLOCK * sizeof(hiprandState)), "Could not allocate memory for randomStates");
	cudasafe(hipMalloc((void **)&devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for devicePopulation");
	cudasafe(hipMalloc((void **)&nextGeneration, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for nextGeneration");
	cudasafe(hipMalloc((void **)&deviceScore, POPULATION_SIZE * sizeof (ScoreWithId)), "Could not allocate memory for deviceScore");

	thrust::device_ptr<ScoreWithId> deviceScorePtrBegin(deviceScore);
	thrust::device_ptr<ScoreWithId> deviceScorePtrEnd = deviceScorePtrBegin + POPULATION_SIZE;

	cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");

	// invoking random init
	randomInit<<<1, MAX_THREADS_PER_BLOCK>>>(randomStates, 900);
	cudasafe(hipGetLastError(), "Could not invoke kernel randomInit");
	cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling randomInit");

	const int BLOCKS_NUMBER = (POPULATION_SIZE + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
	for (int generationIndex=0; generationIndex<30000; ++generationIndex) {
		// invoking calcScore
		calcScore<<<BLOCKS_NUMBER, MAX_THREADS_PER_BLOCK>>>(devicePopulation, deviceScore);
		cudasafe(hipGetLastError(), "Could not invoke kernel calcScore");
		cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calsScore");
		//printPopulation(devicePopulation, deviceScore);

		thrust::sort(deviceScorePtrBegin, deviceScorePtrEnd, ScoreCompare());

		produceGeneration<<<BLOCKS_NUMBER, MAX_THREADS_PER_BLOCK>>>(devicePopulation, nextGeneration, deviceScore, randomStates);
		cudasafe(hipGetLastError(), "Could not invoke kernel produceGeneration");
		cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after produceGeneration");

		std::swap(devicePopulation, nextGeneration);

		if (generationIndex % 1000 == 0) {
			std::cout << "printing first 10 elements of score:" << std::endl;
			cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");
			for (int i=0; i<10; i++)
				std::cout << score[i].score << ' ';
			std::cout << std::endl;
		}
	}

	// freeing memory
	cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");
	cudasafe(hipFree(deviceScore), "Failed to free deviceScore");
	cudasafe(hipFree(randomStates), "Could not free randomStates");
	cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");

	delete[] population;

	return ans;
}

int main() {
	//freopen("output.txt", "w", stdout);
	srand(900);
	srand(static_cast<unsigned>(time(0)));

	double ans = solveGPU();
	std::cout << "GPU answer = " << ans << std::endl;

	return 0;
}