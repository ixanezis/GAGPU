#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <algorithm>
#include <functional>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "constants.h"

// assume block size equal population size

template <class T>
__device__ inline T sqr(const T& value) {
	return value * value;
}

const int THREADS_PER_BLOCK = 256;

void cudasafe(hipError_t error, char* message = "Error occured") {
	if(error != hipSuccess) {
		fprintf(stderr,"ERROR: %s : %i\n", message, error);
		exit(-1);
	}
}

__global__ void randomInit(hiprandState* state, unsigned long seed) {
    int tid = threadIdx.x;
    hiprand_init(seed, tid, 0, state + tid);
}

__device__ float rosenbrock(const float* curPos) {
    float result = 0;
    for (size_t i=0; i<VAR_NUMBER-1; ++i) {
        result += sqr(1 - *curPos) + 100 * sqr(*(curPos+1) - sqr(*curPos));
        ++curPos;
    }
    return result;
}

__device__ float rastrigin(const float *curPos) {
    float result = 10.0f * VAR_NUMBER;
    for (size_t i=0; i<VAR_NUMBER; ++i) {
        result += *curPos * *curPos - 10.0f * cosf(2 * HIP_PI_F * *curPos);
        ++curPos;
    }
    return result;
}

__global__ void GAKernel(float* population, ScoreWithId* score, hiprandState* randomStates) {
	__shared__ float sharedPopulation[THREADS_PER_BLOCK * 2][VAR_NUMBER];
	__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
	const float SIGN[2] = {-1.0f, 1.0f};
    const float MULT[2] = {1.0f, 0.0f};

	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int tid = threadIdx.x;

	// loading initial random population into shared memory
    if (gid < POPULATION_SIZE) {
        for (int i=0; i<VAR_NUMBER; ++i)
            sharedPopulation[tid][i] = population[gid * VAR_NUMBER + i];
    }

    sharedScore[tid + THREADS_PER_BLOCK] = 123123.0;

    __syncthreads();
    // we first have to calculate the score for the first half of threads
    const float *curPos = sharedPopulation[tid];
    sharedScore[tid] = rosenbrock(curPos);

	hiprandState &localState = randomStates[tid];
	for (int generationIndex=0; ; ++generationIndex) {
		__syncthreads();

		// calculating score for the second half of individuals
		const float *curPos = sharedPopulation[tid + THREADS_PER_BLOCK];
		sharedScore[tid + THREADS_PER_BLOCK] = rosenbrock(curPos);

		__syncthreads();

		if (generationIndex == 400000) break;

		// selection
        // first half of threads writes best individual into its position
        if (sharedScore[tid] > sharedScore[tid + THREADS_PER_BLOCK]) {
            for (int i=0; i<VAR_NUMBER; ++i)
                sharedPopulation[tid][i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
            sharedScore[tid] = sharedScore[tid + THREADS_PER_BLOCK];
        }

		__syncthreads();

		// now we've got best individuals in the first half of sharedPopulation

		// crossovers
        const int first = hiprand_uniform(&localState) * THREADS_PER_BLOCK;
        const int second = hiprand_uniform(&localState) * THREADS_PER_BLOCK;
    
        const float weight = hiprand_uniform(&localState);
        for (int i=0; i<VAR_NUMBER; ++i) {
            sharedPopulation[tid + THREADS_PER_BLOCK][i] = sharedPopulation[first][i] * weight + sharedPopulation[second][i] * (1.0f - weight);
        }

		__syncthreads();

		// mutations on second half of population
        if (hiprand_uniform(&localState) < 0.8) {
            const float order = (hiprand_uniform(&localState) * 17) - 15;
            for (int i=0; i<VAR_NUMBER; ++i) {
                const float mult = MULT[hiprand_uniform(&localState) < 0.8f];
                const float sign = SIGN[hiprand_uniform(&localState) < 0.5f];
                const float order_deviation = (hiprand_uniform(&localState) - 0.5f) * 5;
                sharedPopulation[tid + THREADS_PER_BLOCK][i] += powf(10.0f, order + order_deviation) * sign * mult;
            }
        }

        // sharing a part of population with others
        if ((blockIdx.x + generationIndex) % 5 == 0) {
            for (int i=0; i<VAR_NUMBER; ++i)
                population[gid * VAR_NUMBER + i] = sharedPopulation[tid][i];
        }

        // take some best individuals from neighbour
        if ((blockIdx.x + generationIndex) % 3 == 0) {
            if (hiprand_uniform(&localState) < 0.11) {
                const int anotherBlock = hiprand_uniform(&localState) * (POPULATION_SIZE / THREADS_PER_BLOCK);
                const int ngid  = blockDim.x * anotherBlock + threadIdx.x;
                for (int i=0; i<VAR_NUMBER; ++i)
                    sharedPopulation[tid][i] = population[ngid * VAR_NUMBER + i];
                sharedScore[tid] = rosenbrock(sharedPopulation[tid]);
            }
        }
	}

	// output current population back
    if (gid < POPULATION_SIZE) {
        for (int i=0; i<VAR_NUMBER; ++i)
            population[gid * VAR_NUMBER + i] = sharedPopulation[tid][i];

        score[gid].score = sharedScore[tid];
    }
}

void printPopulation(const float* devicePopulation, const ScoreWithId* deviceScore) {
	float population[POPULATION_SIZE][VAR_NUMBER];
	cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

	ScoreWithId score[POPULATION_SIZE];
	cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");

	//std::cout.cetf(std::ios::fixed);
	std::cout.precision(12);
	
	for (int i=0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << i << ' ';
	}
	std::cout << std::endl;

	for (int i=0; i<VAR_NUMBER; i++) {
		for (int u=0; u<POPULATION_SIZE; ++u) {
			std::cout << std::setw(15) << population[u][i] << ' ';
		}
		std::cout << std::endl;
	}
	std::cout << "Score: " << std::endl;
	for (int i=0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << score[i].score << ' ';
	}
	std::cout << std::endl;
}

double solveGPU() {
    cudasafe(hipSetDevice(0), "Could not set device 0");

	double ans = 0;

	float *population = new float[POPULATION_SIZE * VAR_NUMBER];

	for (int i=0; i<POPULATION_SIZE; ++i) {
		for (int u=0; u<VAR_NUMBER; ++u) {
			population[i * VAR_NUMBER + u] = (float_random() - 0.5f) * 10;
		}
	}

	// copying population to device
	float *devicePopulation = 0;
	float *nextGeneration = 0;
	ScoreWithId *deviceScore = 0;
	hiprandState* randomStates;

	cudasafe(hipMalloc(&randomStates, THREADS_PER_BLOCK * sizeof(hiprandState)), "Could not allocate memory for randomStates");
	cudasafe(hipMalloc((void **)&devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for devicePopulation");
	cudasafe(hipMalloc((void **)&nextGeneration, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for nextGeneration");
	cudasafe(hipMalloc((void **)&deviceScore, POPULATION_SIZE * sizeof (ScoreWithId)), "Could not allocate memory for deviceScore");

	cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");

	// invoking random init
	randomInit<<<1, THREADS_PER_BLOCK>>>(randomStates, 900);
	cudasafe(hipGetLastError(), "Could not invoke kernel randomInit");
	cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling randomInit");

	const int BLOCKS_NUMBER = (POPULATION_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    //for (int i=0; i<1115; i++) {
        GAKernel<<<BLOCKS_NUMBER, THREADS_PER_BLOCK>>>(devicePopulation, deviceScore, randomStates);
        cudasafe(hipGetLastError(), "Could not invoke GAKernel");
        cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling GAKernel");

        printPopulation(devicePopulation, deviceScore);
    //}

	// freeing memory
	cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");
	cudasafe(hipFree(deviceScore), "Failed to free deviceScore");
	cudasafe(hipFree(randomStates), "Could not free randomStates");
	cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");

	delete[] population;

	return ans;
}

int main() {
	freopen("output.txt", "w", stdout);
	srand(1900);
	srand(static_cast<unsigned>(time(0)));

	double ans = solveGPU();
	std::cout << "GPU answer = " << ans << std::endl;

	return 0;
}
